#include "hip/hip_runtime.h"
#include "kernel.h"

#include <math.h>

#define GLOBAL_ID ( (((blockDim.y * blockIdx.y) + threadIdx.y) * (gridDim.x * blockDim.x)) + ((blockDim.x * blockIdx.x) + threadIdx.x) )

__global__ void kernel_bgrpacked2rgbapacked(const uchar3* bgr, int w, int h, uchar4* rgba) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = y * w + x;

    if (x < w) {
        rgba[i].x = bgr[i].z;
        rgba[i].y = bgr[i].y;
        rgba[i].z = bgr[i].x;
        rgba[i].w = 255;
    }
}

void convertBGRtoRGBA(uchar4* rgba, const uchar3* bgr, int w, int h, hipStream_t stream) {
    dim3 blocks(512, 1, 1);
    dim3 grid(ceil((float)w / (float)blocks.x), h, 1);

    kernel_bgrpacked2rgbapacked<<<grid, blocks, 0, stream>>>(bgr, w, h, rgba);
}
